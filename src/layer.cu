#include "hip/hip_runtime.h"
/* Last Updated: 24.08.27. 18:30 */
#include "layer.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

/* Linear
 * @param [in1]  in: [M, K]
 * @param [in2]   w: [N, K]
 * @param [in3]   b: [N]
 * @param [out] out: [M, N]
 */
void Linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
  size_t M = out->shape[0];
  size_t N = out->shape[1];
  size_t K = w->shape[1];

  for (size_t m = 0; m < M; m++) {
    for (size_t n = 0; n < N; n++) {
      out->buf[m * N + n] = 0;
      for (size_t k = 0; k < K; k++) {
        out->buf[m * N + n] += in->buf[m * K + k] * w->buf[n * K + k];
      }
      out->buf[m * N + n] += b->buf[n];
    }
  }
}

/* Reshape 
 * @param [in]   in: [N, D]
 * @param [out] out: [N, C, H, W]
 * 'N' is the number of input tensors.
 * 'D' is the dimension of the input tensor.
 * 'C' is the number of channels.
 * 'H' is the height of the output tensor.
 * 'W' is the width of the output tensor.
 */
void Reshape(Tensor *in, Tensor *out) {
  size_t N = in->shape[0];
  size_t D = in->shape[1];
  size_t C = out->shape[1];
  size_t H = out->shape[2];
  size_t W = out->shape[3];

  for (size_t n = 0; n < N; n++) {
    for (size_t c = 0; c < C; c++) {
      for (size_t h = 0; h < H; h++) {
        for (size_t w = 0; w < W; w++) {
          out->buf[n * C * H * W + c * H * W + h * W + w] =
              in->buf[n * D + c * H * W + h * W + w];
        }
      }
    }
  }
}

/* ConvTranspose2d
 * @param [in1]     in: [N, C, H, W]
 * @param [in2] weight: [C, K, R, S]
 * @param [in3]   bias: [K]
 * @param [out]    out: [N, K, OH, OW]
 *    
 *    OH = (H - 1) * stride - 2 * pad + dilation * (R - 1) + output_pad + 1
 *    OW = (W - 1) * stride - 2 * pad + dilation * (S - 1) + output_pad + 1
 *    In this model, R = S = 3, stride = 2, pad = 1, dilation = 1, output_pad = 1
 *
 * 'N' is the number of input tensors.
 * 'C' is the number of input channels.
 * 'H' is the height of the input tensor.
 * 'W' is the width of the input tensor.
 * 'K' is the number of output channels.
 * 'R' is the height of the filter.
 * 'S' is the width of the filter.
 * 'OH' is the height of the output tensor.
 * 'OW' is the width of the output tensor.
 */
void ConvTranspose2d(Tensor *in, Tensor *weight, Tensor *bias, Tensor *out) {
  size_t C = in->shape[1];
  size_t H = in->shape[2];
  size_t W = in->shape[3];
  size_t K = weight->shape[1];
  size_t R = weight->shape[2];
  size_t S = weight->shape[3];
  size_t OH = out->shape[2];
  size_t OW = out->shape[3];
 
  const size_t stride = 2;
  const size_t pad = 1;
  const size_t dilation = 1;

#pragma omp parallel for
  for (size_t oc = 0; oc < K; ++oc) {
    for (size_t oh = 0; oh < OH; ++oh) {
      for (size_t ow = 0; ow < OW; ++ow) {
        half_cpu o = bias->buf[oc];
        for (size_t c = 0; c < C; ++c) {
          for (size_t r = 0; r < R; ++r) {
            for (size_t s = 0; s < S; ++s) {
              if ((oh - (r * dilation - pad)) % stride != 0) continue;
              if ((ow - (s * dilation - pad)) % stride != 0) continue;
              size_t h = (oh - (r * dilation - pad)) / stride;
              size_t w = (ow - (s * dilation - pad)) / stride;
              if (h >= H || w >= W) continue;
              o += in->buf[c * H * W + h * W + w] * 
                weight->buf[c * K * R * S + oc * R * S + r * S + s];
            }
          }
        }
        out->buf[oc * OH * OW + oh * OW + ow] = o;
      }
    }
  }
}

/* BatchNorm2d (track_running_stats=False)
 * @param [in1]     in: [N, C, H, W]
 * @param [in2] weight: [C]
 * @param [in3]   bias: [C]
 * @param [out]    out: [N, C, H, W]  
 * 
 *    out = weight * (in - mean) / sqrt(var + 1e-5) + bias 
 * 
 * 'N' is the number of input tensors.
 * 'C' is the number of channels.
 * 'H' is the height of the input tensor.
 * 'W' is the width of the input tensor.
 */
void BatchNorm2d(Tensor *in, Tensor *weight, Tensor *bias, Tensor *out) {
  size_t C = in->shape[1];
  size_t H = in->shape[2];
  size_t W = in->shape[3];

  const float eps = 1e-5f;

  for (size_t c = 0; c < C; c++) {
    // 1. Caculate mean for each channel
    float mean = 0.0f;
    float var = 0.0f;
    for (size_t h = 0; h < H; h++) {
      for (size_t w = 0; w < W; w++) {
        half_cpu val = in->buf[c * H * W + h * W + w];
        mean += static_cast<float>(val); /* Cast to float */
      }
    }
    mean /= static_cast<float>(H * W);

    // 2. Caculate variance for each channel
    for (size_t h = 0; h < H; h++) {
      for (size_t w = 0; w < W; w++) {
        half_cpu val = in->buf[c * H * W + h * W + w];
        var += (static_cast<float>(val) - mean) * 
          (static_cast<float>(val) - mean); /* Cast to float */
      }
    }
    var /= static_cast<float>(H * W);

    // 3. Normalize with the calculated mean and variance
    for (size_t h = 0; h < H; h++) {
      for (size_t w = 0; w < W; w++) {
        out->buf[c * H * W + h * W + w] =
          weight->buf[c] * 
          (in->buf[c * H * W + h * W + w] - 
          half_cpu(mean)) / /* Cast to half */
          half_cpu(sqrt(var + eps)) + /* Cast to half */
          bias->buf[c];
      }
    }
  }
}

/* LeakyReLU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
void LeakyReLU(Tensor *inout) {
  size_t N = inout->num_elem();

  const half_cpu alpha = 0.01_h;

  for (size_t i = 0; i < N; i++) {
    if (inout->buf[i] < 0) { inout->buf[i] *= alpha; }
  }
}

/* LeakyReLU GPU kernel
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
__global__ void LeakyReLU_kernel(half *inout, size_t N, half alpha) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    if (inout[idx] < half(0)) { inout[idx] *= alpha; }
  }
}

/* LeakyReLU using CUDA GPU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
void LeakyReLU_cuda(Tensor *inout) {
  size_t N = inout->num_elem();

  const half alpha = 0.01;
  
  half *d_inout;

  CHECK_CUDA(hipMalloc(&d_inout, N * sizeof(half)));
  CHECK_CUDA(hipMemcpy(d_inout, inout->buf, N * sizeof(half), hipMemcpyHostToDevice));

  LeakyReLU_kernel<<<(N + 255) / 256, 256>>>(d_inout, N, alpha);
  CHECK_CUDA(hipDeviceSynchronize());

  CHECK_CUDA(hipMemcpy(inout->buf, d_inout, N * sizeof(half), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(d_inout));
}

/* Conv2d
 * @param [in1]     in: [N, C, H, W]
 * @param [in2] weight: [K, C, R, S]
 * @param [in3]   bias: [K]
 * @param [out]    out: [N, K, OH, OW]
 *
 *   OH = (H + 2 * pad - dilation * (R - 1) - 1) / stride + 1
 *   OW = (W + 2 * pad - dilation * (S - 1) - 1) / stride + 1
 *   In this model, R = S = 3, stride = 1, pad = 1, dilation = 1
 *
 * 'N' is the number of input tensors.
 * 'C' is the number of input channels.
 * 'H' is the height of the input tensor.
 * 'W' is the width of the input tensor.
 * 'K' is the number of output channels.
 * 'R' is the height of the filter.
 * 'S' is the width of the filter.
 * 'OH' is the height of the output tensor.
 * 'OW' is the width of the output tensor.
 */
void Conv2d(Tensor *in, Tensor *weight, Tensor *bias, Tensor *out) {
  size_t N = in->shape[0];
  size_t C = in->shape[1];
  size_t H = in->shape[2];
  size_t W = in->shape[3];
  size_t K = weight->shape[0];
  size_t R = weight->shape[2];
  size_t S = weight->shape[3];
  size_t OH = out->shape[2];
  size_t OW = out->shape[3];

  const size_t stride = 1;
  const size_t pad = 1;
  const size_t dilation = 1;

  for (size_t n = 0; n < N; n++) {
    for (size_t oc = 0; oc < K; oc++) {
      for (size_t oh = 0; oh < OH; oh++) {
        for (size_t ow = 0; ow < OW; ow++) {
          half_cpu o = bias->buf[oc];
          for (size_t c = 0; c < C; c++) {
            for (size_t r = 0; r < R; r++) {
              for (size_t s = 0; s < S; s++) {
                size_t h = oh * stride - pad + r * dilation;
                size_t w = ow * stride - pad + s * dilation;
                if (h >= H || w >= W) continue;
                o += in->buf[n * C * H * W + c * H * W + h * W + w] *
                  weight->buf[oc * C * R * S + c * R * S + r * S + s];
              }
            }
          }
          out->buf[n * K * OH * OW + oc * OH * OW + oh * OW + ow] = o;
        }
      }
    }
  }
}

/* Tanh 
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
void Tanh(Tensor *inout) {
  size_t N = inout->num_elem();

  for (size_t i = 0; i < N; i++) {
    inout->buf[i] = tanh(inout->buf[i]);
  }
}

